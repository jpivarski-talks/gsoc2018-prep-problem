#include "hip/hip_runtime.h"
#include "stdio.h"
#include <ctime>
#include <iostream>

__global__ void reduce_a(int lenparents, int d, int* parents, float* mutablescan) {
  unsigned int i = threadIdx.x + blockIdx.x*blockDim.x;
  if (i < lenparents  &&  i >= d  &&  parents[i] == parents[i - d]) {
    mutablescan[i] = mutablescan[i] + mutablescan[i - d];
  }
  // __syncthreads();
}

__global__ void reduce_b(int lenstarts, int* starts, float* scan, float* output) {
  unsigned int i = threadIdx.x + blockIdx.x*blockDim.x;
  if (i < lenstarts) {
    if (offsets[i] == offsets[i + 1]) {
      output[i] = 0.0;
    }
    else {
      output[i] = scan[offsets[i + 1] - 1];
    }
  }
}

int main(int argc, char** argv) {
  FILE *f_offsets = fopen(argv[1], "r");
  fseek(f_offsets, 0, SEEK_END);
  int num_offsets = ftell(f_offsets) / 4;
  fseek(f_offsets, 0, SEEK_SET);
  int* offsets = (int*)malloc(num_offsets * 4);
  fread(offsets, 4, num_offsets, f_offsets);
  
  FILE *f_parents = fopen(argv[2], "r");
  fseek(f_parents, 0, SEEK_END);
  int num_parents = ftell(f_parents) / 4;
  fseek(f_parents, 0, SEEK_SET);
  int* parents = (int*)malloc(num_parents * 4);
  fread(parents, 4, num_parents, f_parents);
  
  FILE *f_content = fopen(argv[3], "r");
  fseek(f_content, 0, SEEK_END);
  int num_content = ftell(f_content) / 4;
  fseek(f_content, 0, SEEK_SET);
  float* content = (float*)malloc(num_content * 4);
  fread(content, 4, num_content, f_content);
  
  int* gpu_offsets;
  int* gpu_parents;
  float* gpu_content;
  float* gpu_output;
  
  hipMalloc((void**)&gpu_offsets, num_offsets * 4);
  hipMalloc((void**)&gpu_parents, num_parents * 4);
  hipMalloc((void**)&gpu_content, num_content * 4);
  hipMalloc((void**)&gpu_output, (num_offsets - 1) * 4);

  float* output = (float*)malloc((num_offsets - 1) * 4);

  int numtimes = 10;
  double totaltime = 0.0;
  for (int time = 0;  time < numtimes;  time++) {
    hipMemcpy(gpu_offsets, offsets, num_offsets * 4, hipMemcpyHostToDevice);
    hipMemcpy(gpu_parents, parents, num_parents * 4, hipMemcpyHostToDevice);
    hipMemcpy(gpu_content, content, num_content * 4, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    int threadsperblock = 1024;
    int blocksize = num_parents / threadsperblock + 1;

    std::clock_t starttime = std::clock();

    for (int d = 1;  d < num_parents;  d *= 2) {
      reduce_a<<<blocksize, threadsperblock>>>(num_parents, d, gpu_parents, gpu_content);
      hipDeviceSynchronize();
    }

    blocksize = (num_offsets - 1) / threadsperblock + 1;
    reduce_b<<<blocksize, threadsperblock>>>(num_offsets - 1, gpu_offsets, gpu_content, gpu_output);

    std::clock_t stoptime = std::clock();

    hipMemcpy(output, gpu_output, (num_offsets - 1) * 4, hipMemcpyDeviceToHost);

    totaltime += (stoptime - starttime) / (double)CLOCKS_PER_SEC;
  }

  std::cout << (1e3 * totaltime) / numtimes << " ms" << std::endl;
}

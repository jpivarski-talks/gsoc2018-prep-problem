
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <ctime>
#include <iostream>

__global__ void reduce_a(int lenparents, int* parents, float* mutablescan) {
  unsigned int i = threadIdx.x + blockIdx.x*blockDim.x;

  for (int d = 1;  d < 1024;  d *= 2) {
    if (i < lenparents  &&  i >= d  &&  parents[i] == parents[i - d]) {
      mutablescan[i] = mutablescan[i] + mutablescan[i - d];
    }
    __syncthreads();
  }
}

__global__ void reduce_b(int lenparents, int* parents, float* mutablescan) {
  unsigned int i = (threadIdx.x + blockIdx.x*blockDim.x + 1) * 1024;
  
  int extra = 0;
  while (i + extra < lenparents  &&  extra < 1024  &&  parents[i + extra] == parents[i - 1]) {
    mutablescan[i + extra] += mutablescan[i - 1];
    extra++;
  }
}

__global__ void reduce_c(int lenstarts, int* offsets, float* scan, float* output) {
  unsigned int i = threadIdx.x + blockIdx.x*blockDim.x;

  if (i < lenstarts) {
    if (offsets[i] == offsets[i + 1]) {
      output[i] = 0.0;
    }
    else {
      output[i] = scan[offsets[i + 1] - 1];
    }
  }
}

int main(int argc, char** argv) {
  FILE *f_offsets = fopen(argv[1], "r");
  fseek(f_offsets, 0, SEEK_END);
  int num_offsets = ftell(f_offsets) / 4;
  fseek(f_offsets, 0, SEEK_SET);
  int* offsets = (int*)malloc(num_offsets * 4);
  fread(offsets, 4, num_offsets, f_offsets);
  
  FILE *f_parents = fopen(argv[2], "r");
  fseek(f_parents, 0, SEEK_END);
  int num_parents = ftell(f_parents) / 4;
  fseek(f_parents, 0, SEEK_SET);
  int* parents = (int*)malloc(num_parents * 4);
  fread(parents, 4, num_parents, f_parents);
  
  FILE *f_content = fopen(argv[3], "r");
  fseek(f_content, 0, SEEK_END);
  int num_content = ftell(f_content) / 4;
  fseek(f_content, 0, SEEK_SET);
  float* content = (float*)malloc(num_content * 4);
  fread(content, 4, num_content, f_content);
  
  int* gpu_offsets;
  int* gpu_parents;
  float* gpu_content;
  float* gpu_output;
  
  hipMalloc((void**)&gpu_offsets, num_offsets * 4);
  hipMalloc((void**)&gpu_parents, num_parents * 4);
  hipMalloc((void**)&gpu_content, num_content * 4);
  hipMalloc((void**)&gpu_output, (num_offsets - 1) * 4);

  float* output = (float*)malloc((num_offsets - 1) * 4);

  int numtimes = 10;
  double totaltime = 0.0;
  for (int time = 0;  time < numtimes;  time++) {
    std::clock_t starttime = std::clock();

    hipMemcpy(gpu_offsets, offsets, num_offsets * 4, hipMemcpyHostToDevice);
    hipMemcpy(gpu_parents, parents, num_parents * 4, hipMemcpyHostToDevice);
    hipMemcpy(gpu_content, content, num_content * 4, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    int threadsperblock = 1024;
    int blocksize = num_parents / threadsperblock + 1;

    reduce_a<<<blocksize, threadsperblock>>>(num_parents, gpu_parents, gpu_content);

    blocksize = (num_parents / 1024) / threadsperblock + 1;
    reduce_b<<<blocksize, threadsperblock>>>(num_parents, gpu_parents, gpu_content);

    blocksize = (num_offsets - 1) / threadsperblock + 1;
    reduce_c<<<blocksize, threadsperblock>>>(num_offsets - 1, gpu_offsets, gpu_content, gpu_output);

    hipMemcpy(output, gpu_output, (num_offsets - 1) * 4, hipMemcpyDeviceToHost);
    std::clock_t stoptime = std::clock();

    totaltime += (stoptime - starttime) / (double)CLOCKS_PER_SEC;
  }

  std::cout << argv[3] << " " << (num_content * numtimes) * 1e-6 / totaltime << " MHz" << std::endl;
}
